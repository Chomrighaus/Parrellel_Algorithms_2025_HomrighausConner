// Name: Conner Homrighaus
// Vector Dot product on 1 block 
// nvcc HW8.cu -o run
/*
 What to do:
 This code uses the CPU to compute the dot product of two vectors of length N. 
 It includes a skeleton for setting up a GPU dot product, but that part is currently empty.
 Additionally, the CPU code is somewhat convoluted, but it is structured this way to parallel 
 the GPU code you will need to write. The program will also verify whether you have correctly 
 implemented the dot product on the GPU.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

// Defines
#define N 2000 // Length of the vector

// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
float DotCPU, DotGPU;
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid
float Tolerance = 0.01;

// Function prototypes
void cudaErrorCheck(const char *, int);
void setUpDevices();
void allocateMemory();
void innitialize();
void dotProductCPU(float*, float*, int);
__global__ void dotProductGPU(float*, float*, float*, int);
bool  check(float, float, float);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
	BlockSize.x = 256;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = 1;
	GridSize.y = 1;
	GridSize.z = 1;
}

// Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
	
	// Device "GPU" Memory
	hipMalloc(&A_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
}

// Loading values into the vectors that we will add.
void innitialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(2*i);
	}
}

// Adding vectors a and b on the CPU then stores result in vector c.
void dotProductCPU(float *a, float *b, float *C_CPU, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		C_CPU[id] = a[id] * b[id];
	}
	
	for(int id = 1; id < n; id++)
	{ 
		C_CPU[0] += C_CPU[id];
	}
}

// This is the kernel. It is the function that will run on the GPU.
// It adds vectors a and b on the GPU then stores result in vector c.
__global__ void dotProductGPU(float *a, float *b, float *C_GPU, int n)
{
	// __shared__ puts the array of 256. However, it can not be dynamic, it is static. So if 
	// more blocks were to be used I would have an issue. However, I am in luck! After some more
	// searching through documentation, I found extern! 
	// extern __shared__ float temp[]; at least thats what I want to say, unfortunately I don't understand
	// what the heck extern is doing, so I guess I will take the hit and simply use the static 256
	__shared__ float temp[256];
	
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Global Index Of a thread! 
	// I am finally starting to grasp how all of these indexes work! (said at 3:08 after a lot of experiementation btw... I have restarted 3 times)
	// Then I realied I was just WAAAAAAYYYYYYY overcomplicating it. I did a lot of silly things... But it is finally running with a small percent error!!!!
    int stride = blockDim.x * gridDim.x; // Stride! 

	// Notice that I used blockIdx.x. That is in case one of the tests we run is to see if adding more blocks/grids will break it!

	// Using temp
    temp[threadIdx.x] = 0;
    while (tid < n)
    {
        temp[threadIdx.x] += a[tid] * b[tid];
        tid += stride;
    }

    __syncthreads(); // Make the threads wait for each other!


	// Everything I was reading online was telling me that when you store the result you want to use
	// threadIdx.x == 0? Even on multiple question boards it was telling me to do this. I was originally
	// struggling to get my code to work, but then I found this and it started actually running again!
    if (threadIdx.x == 0)
    {
		// I decided to store all of the sums together by taking temp and calculating 
		// the sum using sumOfTemp! This will act as my temporary sum, which I will then use
		// to store onto C_GPU
        float sumOfTemp = 0;
        for (int i = 0; i < blockDim.x; i++)
        {
            sumOfTemp += temp[i];
        }
		C_GPU[0] = sumOfTemp;
    }

}
	

// Checking to see if anything went wrong in the vector addition.
bool check(float cpuAnswer, float gpuAnswer, float tolerence)
{
	double percentError;
	
	percentError = abs((gpuAnswer - cpuAnswer)/(cpuAnswer))*100.0;
	printf("\n\n percent error = %lf\n", percentError);
	
	if(percentError < Tolerance) 
	{
		return(true);
	}
	else 
	{
		return(false);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

// Cleaning up memory after we are finished.
void CleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU); 
	free(B_CPU); 
	free(C_CPU);
	
	hipFree(A_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
}

int main()
{
	timeval start, end;
	long timeCPU, timeGPU;
	
	// Setting up the GPU
	setUpDevices();
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	innitialize();
	
	// Adding on the CPU
	gettimeofday(&start, NULL);
	dotProductCPU(A_CPU, B_CPU, C_CPU, N);
	DotCPU = C_CPU[0];
	gettimeofday(&end, NULL);
	timeCPU = elaspedTime(start, end);
	
	// Adding on the GPU
	gettimeofday(&start, NULL);
	
	// Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	dotProductGPU<<<GridSize,BlockSize>>>(A_GPU, B_GPU, C_GPU, N);
	cudaErrorCheck(__FILE__, __LINE__);

	// Making sure the GPU and CPU wiat until each other are at the same place.
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	DotGPU = C_CPU[0]; // C_GPU was copied into C_CPU.

	gettimeofday(&end, NULL);
	timeGPU = elaspedTime(start, end);
	
	// Checking to see if all went correctly.
	if(check(DotCPU, DotGPU, Tolerance) == false)
	{
		printf("\n\n Something went wrong in the GPU dot product.\n");
	}
	else
	{
		printf("\n\n You did a dot product correctly on the GPU");
		printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
		printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
	}
	
	// Your done so cleanup your room.	
	CleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n\n");
	
	return(0);
}



#include "hip/hip_runtime.h"
// Name: Conner Homrighaus
// Creating a GPU nBody simulation from an nBody CPU simulation. 
// nvcc HW18.cu -o temp -lglut -lm -lGLU -lGL

/*
 What to do:
 This is some lean nBody code that runs on the CPU. Rewrite it, keeping the same general format, 
 but offload the compute-intensive parts of the code to the GPU for acceleration.
 Note: The code takes two arguments as inputs:
 1. The number of bodies to simulate, (We will keep the number of bodies under 1024 for this HW so it can be run on one block.)
 2. Whether to draw sub-arrangements of the bodies during the simulation (1), or only the first and last arrangements (0).
*/

// Include files
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Defines
#define PI 3.14159265359
#define DRAW_RATE 10

// This is to create a Lennard-Jones type function G/(r^p) - H(r^q). (p < q) p has to be less than q.
// In this code we will keep it a p = 2 and q = 4 problem. The diameter of a body is found using the general
// case so it will be more robust but in the code leaving it as a set 2, 4 problem make the coding much easier.
#define G 10.0
#define H 10.0
#define LJP  2.0
#define LJQ  4.0

#define DT 0.0001
#define RUN_TIME 1.0

// Globals
int N, DrawFlag;
float3 *P, *V, *F;
float3 *P_GPU, *V_GPU, *F_GPU;
float *M, *M_GPU; 
float GlobeRadius, Diameter, Radius;
float Damp;
dim3 BlockSize, GridSize;

// Function prototypes
void setupCUDA();
void allocateMemCud();
void freeAll();
__global__ void forcesCUDA(float3*, float3*, float3*, float*, int);
__global__ void positionsCUDA(float3*, float3*, float3*, float*, int, float, float, float);
void keyPressed(unsigned char, int, int);
long elaspedTime(struct timeval, struct timeval);
void drawPicture();
void timer();
void setup();
void nBody();
void cudaErrorCheck(const char*, int);
int main(int, char**);

// cuda error!!! Don't forget it! 
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// Set up CUDA
void setupCUDA()
{
	BlockSize.x = N;
	BlockSize.y = 1;
	BlockSize.z = 1;

	GridSize.x = 1;
	GridSize.y = 1;
	GridSize.z = 1;

	if(N > 1024)
	{
		printf("\n The number of bodies is greater than 1024. Exiting.\n");
		exit(0);
	}
}

// Allocate Memory I need
void allocateMemCud()
{
    hipMalloc(&P_GPU, N*sizeof(float3));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&V_GPU, N*sizeof(float3));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&F_GPU, N*sizeof(float3));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&M_GPU, N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
}
// Calculate Forces!!!
__global__ void forcesCUDA(float3* p, float3* v, float3* f, float* m, int n)
{
	// INDEX
	int id = threadIdx.x + blockIdx.x*blockDim.x;

    /* OG Nbody Force!!!
    for(int i=0; i<N; i++)
		{
			F[i].x = 0.0;
			F[i].y = 0.0;
			F[i].z = 0.0;
		}
		for(int i=0; i<N; i++)
		{
			for(int j=i+1; j<N; j++)
			{
				dx = P[j].x-P[i].x;
				dy = P[j].y-P[i].y;
				dz = P[j].z-P[i].z;
				d2 = dx*dx + dy*dy + dz*dz;
				d  = sqrt(d2);
				
				force_mag  = (G*M[i]*M[j])/(d2) - (H*M[i]*M[j])/(d2*d2);
				F[i].x += force_mag*dx/d;
				F[j].x -= force_mag*dx/d;
				F[i].y += force_mag*dy/d;
				F[j].y -= force_mag*dy/d;
				F[i].z += force_mag*dz/d;
				F[j].z -= force_mag*dz/d;
			}
		}
    */
	
	if(id < n)
	{
		// Force magnitude 
		float force_mag;
        // Distances
		float dx, dy, dz, d, d2;

		// Start with all forces initalized to 0
		f[id].x = 0.0;
		f[id].y = 0.0;
		f[id].z = 0.0;

		// Loop through all the other bodies and get the force on the body we are working on.
		for(int i = 0; i < n; i++)
		{
			// Make sure we don't grab the force of the body we are currently on!
			if(i != id)
			{
				// Find the distance between the two bodies
				dx = p[i].x - p[id].x;
				dy = p[i].y - p[id].y;
				dz = p[i].z - p[id].z;
                d2 = dx*dx + dy*dy + dz*dz;
				d = sqrt(d2);

				// Calculate the force magnitude.
				force_mag = (G*m[id]*m[i])/(d2) - (H*m[id]*m[i])/(d2*d2);
				// Add the force to the body we are working on.
				f[id].x += force_mag*dx/d;
				f[id].y += force_mag*dy/d;
				f[id].z += force_mag*dz/d;
			}
		}
	}
}

__global__ void positionsCUDA(float3* p, float3* v, float3* f, float* m, int n, float dt, float time, float damp)
{
	// This kernel will update the position of each body :D
	
    // Index!!!
	int id = threadIdx.x + blockIdx.x*blockDim.x;

    /* Position Math From OG Nbody
    for(int i=0; i<N; i++)
		{
			if(time == 0.0)
			{
				V[i].x += (F[i].x/M[i])*0.5*dt;
				V[i].y += (F[i].y/M[i])*0.5*dt;
				V[i].z += (F[i].z/M[i])*0.5*dt;
			}
			else
			{
				V[i].x += ((F[i].x-Damp*V[i].x)/M[i])*dt;
				V[i].y += ((F[i].y-Damp*V[i].y)/M[i])*dt;
				V[i].z += ((F[i].z-Damp*V[i].z)/M[i])*dt;
			}
			P[i].x += V[i].x*dt;
			P[i].y += V[i].y*dt;
			P[i].z += V[i].z*dt;
		}
    */
	
	if(id < n)
	{
		// Update velocity and position!
		if(time == 0.0)
		{
			v[id].x += (f[id].x/m[id])*0.5*dt;
			v[id].y += (f[id].y/m[id])*0.5*dt;
			v[id].z += (f[id].z/m[id])*0.5*dt;
		}
		else
		{
			v[id].x += ((f[id].x-damp*v[id].x)/m[id])*dt;
			v[id].y += ((f[id].y-damp*v[id].y)/m[id])*dt;
			v[id].z += ((f[id].z-damp*v[id].z)/m[id])*dt;
		}
		p[id].x += v[id].x*dt;
		p[id].y += v[id].y*dt;
		p[id].z += v[id].z*dt;
	}
    // I tried to follow along using the previous nbody... Might need to get some
    // explained here. I'm pretty confident I somewhat understand, but I would
    // really appreciate maybe going over this in the future.
    // I think I will work this out as just an nbody problem on paper... I think that would help.
    // That reminds me, I need to go back over my adjacency matrix.
}

void keyPressed(unsigned char key, int x, int y)
{
	if(key == 's')
	{
		timer();
	}
	
	if(key == 'q')
	{
		// Free EVERYTHING! We need to clean up the room!!!
        freeAll();
		exit(0);
	}
}

void freeAll()
{
    free(M);
	free(P);
	free(V);
	free(F);
	hipFree(P_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(V_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(F_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(M_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

void drawPicture()
{
	int i;
	
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);

    // I have some time so I'm gonna make it pretty! 
    // My favorite gradiant time!!!
	for(i = 0; i < N; i++)
	{
		// Calculate a gradient factor between 0 and 1
		float gradientFactor = (float)i / (float)(N - 1);
		
		// Color ranging from blue (0.0, 0.0, 1.0) to green (0.0, 1.0, 0.0)
		glColor3f(0.0, gradientFactor, 1.0 - gradientFactor);
		
		glPushMatrix();
		glTranslatef(P[i].x, P[i].y, P[i].z);
		glutSolidSphere(Radius, 20, 20);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}

void timer()
{	
	timeval start, end;
	long computeTime;
	
    // The way this was not all on the same line really messed
    // with me so I fixed it.
	drawPicture();
	gettimeofday(&start, NULL);
    nBody();
    gettimeofday(&end, NULL);
    drawPicture();
    	
	computeTime = elaspedTime(start, end);
	printf("\n The compute time was %ld microseconds.\n\n", computeTime);
}

void setup()
{
    	float randomAngle1, randomAngle2, randomRadius;
    	float d, dx, dy, dz;
    	int test;
    	
    	Damp = 0.5;
    	
    	M = (float*)malloc(N*sizeof(float));
    	P = (float3*)malloc(N*sizeof(float3));
    	V = (float3*)malloc(N*sizeof(float3));
    	F = (float3*)malloc(N*sizeof(float3));
    	
	
	Diameter = pow(H/G, 1.0/(LJQ - LJP)); // This is the value where the force is zero for the L-J type force.
	Radius = Diameter/2.0;
	
	// Using the radius of a body and a 68% packing ratio to find the radius of a global sphere that should hold all the bodies.
	// Then we double this radius just so we can get all the bodies setup with no problems. 
	float totalVolume = float(N)*(4.0/3.0)*PI*Radius*Radius*Radius;
	totalVolume /= 0.68;
	float totalRadius = pow(3.0*totalVolume/(4.0*PI), 1.0/3.0);
	GlobeRadius = 2.0*totalRadius;
	
	// Randomly setting these bodies in the glaobal sphere and setting the initial velosity, inotial force, and mass.
	for(int i = 0; i < N; i++)
	{
		test = 0;
		while(test == 0)
		{
			// Get random position.
			randomAngle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			randomAngle2 = ((float)rand()/(float)RAND_MAX)*PI;
			randomRadius = ((float)rand()/(float)RAND_MAX)*GlobeRadius;
			P[i].x = randomRadius*cos(randomAngle1)*sin(randomAngle2);
			P[i].y = randomRadius*sin(randomAngle1)*sin(randomAngle2);
			P[i].z = randomRadius*cos(randomAngle2);
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				dx = P[i].x-P[j].x;
				dy = P[i].y-P[j].y;
				dz = P[i].z-P[j].z;
				d = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < Diameter)
				{
					test = 0;
					break;
				}
			}
		}
	
		V[i].x = 0.0;
		V[i].y = 0.0;
		V[i].z = 0.0;
		
		F[i].x = 0.0;
		F[i].y = 0.0;
		F[i].z = 0.0;
		
		M[i] = 1.0;
	}
	setupCUDA();
    allocateMemCud();
	printf("\n To start timing type s.\n");
}

/* OG nBody 
void nBody()
{
	float force_mag; 
	float dx,dy,dz,d, d2;
	int    drawCount = 0; 
	float  time = 0.0;
	float dt = 0.0001;
	while(time < RUN_TIME)
	{
		for(int i=0; i<N; i++)
		{
			F[i].x = 0.0;
			F[i].y = 0.0;
			F[i].z = 0.0;
		}
		for(int i=0; i<N; i++)
		{
			for(int j=i+1; j<N; j++)
			{
				dx = P[j].x-P[i].x;
				dy = P[j].y-P[i].y;
				dz = P[j].z-P[i].z;
				d2 = dx*dx + dy*dy + dz*dz;
				d  = sqrt(d2);
				force_mag  = (G*M[i]*M[j])/(d2) - (H*M[i]*M[j])/(d2*d2);
				F[i].x += force_mag*dx/d;
				F[j].x -= force_mag*dx/d;
				F[i].y += force_mag*dy/d;
				F[j].y -= force_mag*dy/d;
				F[i].z += force_mag*dz/d;
				F[j].z -= force_mag*dz/d;
			}
		}
		for(int i=0; i<N; i++)
		{
			if(time == 0.0)
			{
				V[i].x += (F[i].x/M[i])*0.5*dt;
				V[i].y += (F[i].y/M[i])*0.5*dt;
				V[i].z += (F[i].z/M[i])*0.5*dt;
			}
			else
			{
				V[i].x += ((F[i].x-Damp*V[i].x)/M[i])*dt;
				V[i].y += ((F[i].y-Damp*V[i].y)/M[i])*dt;
				V[i].z += ((F[i].z-Damp*V[i].z)/M[i])*dt;
			}
			P[i].x += V[i].x*dt;
			P[i].y += V[i].y*dt;
			P[i].z += V[i].z*dt;
		}
		if(drawCount == DRAW_RATE) 
		{
			if(DrawFlag) drawPicture();
			drawCount = 0;
		}
		time += dt;
		drawCount++;
	}
}
*/
void nBody()
{
	int    drawCount = 0; 
	float  time = 0.0;
	float dt = 0.0001;

	// Copying the data to the GPU.
	hipMemcpyAsync(P_GPU, P, N*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(V_GPU, V, N*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(F_GPU, F, N*sizeof(float3), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(M_GPU, M, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);

	while(time < RUN_TIME)
	{
		// Calculating the forces for each body.
		forcesCUDA<<<GridSize, BlockSize>>>(P_GPU, V_GPU, F_GPU, M_GPU, N);
		cudaErrorCheck(__FILE__, __LINE__);

		// Updating the position of each body.
		positionsCUDA<<<GridSize, BlockSize>>>(P_GPU, V_GPU, F_GPU, M_GPU, N, dt, time, Damp);
		cudaErrorCheck(__FILE__, __LINE__);

        /*
        if(drawCount == DRAW_RATE) 
		{
			if(DrawFlag) drawPicture();
			drawCount = 0;
		}
        */
		if(drawCount == DRAW_RATE) 
		{
			if(DrawFlag)
			{
				// Copying the data back to the CPU.
				// hipMemcpy acts as a synchronization point.
				hipMemcpy(P, P_GPU, N*sizeof(float3), hipMemcpyDeviceToHost);
				cudaErrorCheck(__FILE__, __LINE__);
				drawPicture();
			}
			drawCount = 0;
		}
		/*
        time += dt;
		drawCount++;
        */
		time += dt;
		drawCount++;
	}
	// Copying the data back to the CPU one last time.
	// hipMemcpy acts as a synchronization point.
	hipMemcpy(P, P_GPU, N*sizeof(float3), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
}

int main(int argc, char** argv)
{
	if( argc < 3)
	{
		printf("\n You need to enter the number of bodies (an int)"); 
		printf("\n and if you want to draw the bodies as they move (1 draw, 0 don't draw),");
		printf("\n on the comand line.\n"); 
		exit(0);
	}
	else
	{
		N = atoi(argv[1]);
		DrawFlag = atoi(argv[2]);
	}
	
	setup();
	
	int XWindowSize = 1000;
	int YWindowSize = 1000;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("nBody Test");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutKeyboardFunc(keyPressed);
	glutDisplayFunc(drawPicture);
	
	float3 eye = {0.0f, 0.0f, 2.0f*GlobeRadius};
	float near = 0.2;
	float far = 5.0*GlobeRadius;
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, near, far);
	glMatrixMode(GL_MODELVIEW);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	gluLookAt(eye.x, eye.y, eye.z, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	
	glutMainLoop();
	return 0;
}
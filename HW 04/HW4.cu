// Name: Conner Homrighaus
// nvcc HW4.cu -o temp
/*
 What to do:
 This is the solution to HW3. It works well for adding vectors with fixed-size blocks. 
 Given the size of the vector it needs to add, it takes a set block size, determines how 
 many blocks are needed, and creates a grid large enough to complete the task. Cool, cool!
 
 But—and this is a big but—this can get you into trouble because there is a limited number 
 of blocks you can use. Though large, it is still finite. Therefore, we need to write the 
 code in such a way that we don't have to worry about this limit. Additionally, some block 
 and grid sizes work better than others, which we will explore when we look at the 
 streaming multiprocessors.
 
 Extend this code so that, given a block size and a grid size, it can handle any vector addition. 
 Start by hard-coding the block size to 256 and the grid size to 64. Then, experiment with different 
 block and grid sizes to see if you can achieve any speedup. Set the vector size to a very large value 
 for time testing.

 You’ve probably already noticed that the GPU doesn’t significantly outperform the CPU. This is because 
 we’re not asking the GPU to do much work, and the overhead of setting up the GPU eliminates much of the 
 potential speedup. 
 
 To address this, modify the computation so that:
 c = sqrt(cos(a)*cos(a) + a*a + sin(a)*sin(a) - 1.0) + sqrt(cos(b)*cos(b) + b*b + sin(b)*sin(b) - 1.0)
 Hopefully, this is just a convoluted and computationally expensive way to calculate a + b.
 If the compiler doesn't recognize the simplification and optimize away all the unnecessary work, 
 this should create enough computational workload for the GPU to outperform the CPU.

 Write the loop as a for loop rather than a while loop. This will allow you to also use #pragma unroll 
 to explore whether it provides any speedup. Make sure to include an if (id < n) condition in your code 
 to ensure safety. Finally, be prepared to discuss the impact of #pragma unroll and whether it helped 
 improve performance.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <math.h>

// Defines
#define N 11504 // Length of the vector

// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid
float Tolerance = 0.00000001;

// Function prototypes
void setUpDevices();
void allocateMemory();
void innitialize();
void addVectorsCPU(float*, float*, float*, int);
__global__ void addVectorsGPU(float, float, float, int);
int  check(float*, int);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
	BlockSize.x = 256;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = 64; 
	GridSize.y = 1;
	GridSize.z = 1;
}

// Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
	
	// Device "GPU" Memory
	hipMalloc(&A_GPU,N*sizeof(float));
	hipMalloc(&B_GPU,N*sizeof(float));
	hipMalloc(&C_GPU,N*sizeof(float));

}

// Loading values into the vectors that we will add.
void innitialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(2*i);
	}
}

// Adding vectors a and b on the CPU then stores result in vector c.
void addVectorsCPU(float *a, float *b, float *c, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		c[id] = sqrt(a[id]*a[id] + (sin(a[id])*sin(a[id]) + cos(a[id])*cos(a[id]) - 1)) + sqrt(b[id]*b[id] + (sin(b[id])*sin(b[id]) + cos(b[id])*cos(b[id]) - 1));
	}
}

// This is the kernel. It is the function that will run on the GPU.
// It adds vectors a and b on the GPU then stores result in vector c.
__global__ void addVectorsGPU(float *a, float *b, float *c, int n)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;
	
	if(id < N)
		{
			#pragma unroll 2
			for (int i = id; i < n; i += stride)
        	{
				c[id] = sqrt(a[id]*a[id] + (sin(a[id])*sin(a[id]) + cos(a[id])*cos(a[id]) - 1)) + sqrt(b[id]*b[id] + (sin(b[id])*sin(b[id]) + cos(b[id])*cos(b[id]) - 1));
        	}
		}

}


// Checking to see if anything went wrong in the vector addition.
int check(float *c, int n)
{
	double sum = 0.0;
	double m = n-1; // Needed the -1 because we start at 0.
	
	for(int id = 0; id < n; id++)
	{ 
		sum += c[id];
	}
	
	if(abs(sum - 3.0*(m*(m+1))/2.0) < Tolerance) 
	{
		return(1);
	}
	else 
	{
		return(0);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

// Cleaning up memory after we are finished.
void CleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU); 
	free(B_CPU); 
	free(C_CPU);
	
	hipFree(A_GPU); 
	hipFree(B_GPU); 
	hipFree(C_GPU);
}

int main()
{
	timeval start, end;
	long timeCPU, timeGPU;
	
	// Setting up the GPU
	setUpDevices();
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	innitialize();
	
	// Adding on the CPU
	gettimeofday(&start, NULL);
	addVectorsCPU(A_CPU, B_CPU ,C_CPU, N);

	gettimeofday(&end, NULL);
	timeCPU = elaspedTime(start, end);
	// Zeroing out the C_CPU vector just to be safe because right now it has the correct answer in it.
	for(int id = 0; id < N; id++)
	{ 
		C_CPU[id] = 0.0;
	}
	
	// Adding on the GPU
	gettimeofday(&start, NULL);
	
	// Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	
	addVectorsGPU<<<GridSize,BlockSize>>>(A_GPU, B_GPU ,C_GPU, N);

	// Making sure the GPU and CPU wait until each other are at the same place.
	hipDeviceSynchronize();

	// Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);

	gettimeofday(&end, NULL);
	timeGPU = elaspedTime(start, end);
	
	// Checking to see if all went correctly.
	if(check(C_CPU, N) == 0)
	{
		printf("\n\n Something went wrong in the GPU vector addition\n");
	}
	else
	{
		printf("\n\n You added the two vectors correctly on the GPU");
		printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
		printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
	}
	
	// Your done so cleanup your room.	
	CleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n\n");
	
	return(0);
}


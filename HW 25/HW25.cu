#include "hip/hip_runtime.h"
// Name: Conner Homrighaus
// nBody run on all available GPUs. 
// nvcc HW25.cu -o temp -lglut -lm -lGLU -lGL

/*
 What to do:
 This is some robust N-body code with all the bells and whistles removed. 
 It runs on two GPUs and two GPUs only. Rewrite it so it automatically detects the number of 
 available GPUs on the machine and runs using all of them.
*/

// Include files
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Defines
#define BLOCK_SIZE 128
#define PI 3.14159265359
#define DRAW_RATE 10

// This is to create a Lennard-Jones type function G/(r^p) - H(r^q). (p < q) p has to be less than q.
// In this code we will keep it a p = 2 and q = 4 problem. The diameter of a body is found using the general
// case so it will be more robust but in the code leaving it as a set 2, 4 problem make the coding much easier.
#define G 10.0f
#define H 10.0f
#define LJP  2.0
#define LJQ  4.0

#define DT 0.0001
#define RUN_TIME 1.0


// Globals
int N;
int *BGPU; // Bodies on each GPU
int NumberOfGpus; // This will hold the number of gpus
float3 *P, *V, *F;
float *M; 
// I need to use array of pointers. 
float3 **PGPU[25], **VGPU[25], **FGPU[25];
float **MGPU[25];
float GlobeRadius, Diameter, Radius;
float Damp;
dim3 BlockSize;
dim3 *GridSize;

// Function prototypes
void cudaErrorCheck(const char *, int);
void drawPicture();
void setup(int NumberOfGpus);
__global__ void getForces(float3 *, float3 *, float3 *, float *, float, float, int, int, int);
__global__ void moveBodies(float3 *, float3 *, float3 *, float *, float, float, float, int, int, int);
void nBody(int NumberOfGpus);
void cleanUpRoom(int NumberOfGpus);
int main(int, char**);

void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

void drawPicture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	hipSetDevice(0);
	hipMemcpyAsync(P, PGPU[0], N*sizeof(float3), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	
	glColor3d(1.0,1.0,0.5);
	
	for(int i=0; i<N; i++)
	{
		glPushMatrix();
		glTranslatef(P[i].x, P[i].y, P[i].z);
		glutSolidSphere(Radius,20,20);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}

void setup()
{
    float randomAngle1, randomAngle2, randomRadius;
    float d, dx, dy, dz;
    int test;

	// This will set up the number of bodies 
	N = 101;
	
	// Get the number of GPUS
	hipGetDeviceCount(&NumberOfGpus);
	if(NumberOfGpus == 0) // No GPUS
	{
		printf("\n Initiating cashapp, trust me you need it. Get a GPU broke human!\n");
		exit(0);
	}
	else // uncountable? negative? matter subtracted from the univers?
	{ 
		printf("\n Dude... How? Just how? I don't even think Elon Musk with all of his money could invent negative gpus...\n");
		printf("\n I won't share my code or play with liars!\n");
		exit(0);
	}
	printf("\n The number of GPUs is: %d\n", NumberOfGpus);

	// Allocate memory for arrays dependent on the number of GPUs
	BGPU = (int*)malloc(NumberOfGpus*sizeof(int));
	PGPU = (float3**)malloc(NumberOfGpus*sizeof(float3*));
	VGPU = (float3**)malloc(NumberOfGpus*sizeof(float3*));
	FGPU = (float3**)malloc(NumberOfGpus*sizeof(float3*));
	MGPU = (float**)malloc(NumberOfGpus*sizeof(float*));
	GridSizes = (dim3*)malloc(NumberOfGpus*sizeof(dim3));

	// Now we set up the memory on CPU.
    M = (float*)malloc(N*sizeof(float));
    P = (float3*)malloc(N*sizeof(float3));
    V = (float3*)malloc(N*sizeof(float3));
    F = (float3*)malloc(N*sizeof(float3));
	
	// Now lets set up the number of bodies on each and every gpu
	for(int i = 0; i < NumberOfGpus; i++)
	{
		BGPU[i] = N/NumberOfGpus;
	}
	BGPU[NumberOfGpus-1] += N % NumberOfGpus; // This is to make sure we have all the bodies accounted for.
	
	// Set up the blocks
	BlockSize.x = 128;
	BlockSize.y = 1;
	BlockSize.z = 1;
		
	// I really liked Kyle's multiple number of gpus gridsize! So I am going to make sure that I get this!
	for(int i = 0; i < NumberOfGpus; i++)
	{
		GridSizes[i].x = (NGPUs[i] - 1)/BlockSize.x + 1;
		GridSizes[i].y = 1;
		GridSizes[i].z = 1;
	}
	
	Diameter = pow(H/G, 1.0/(LJQ - LJP)); // This is the value where the force is zero for the L-J type force.
	Radius = Diameter/2.0;
	
	// Using the radius of a body and a 68% packing ratio to find the radius of a global sphere that should hold all the bodies.
	// Then we double this radius just so we can get all the bodies setup with no problems. 
	float totalVolume = float(N)*(4.0/3.0)*PI*Radius*Radius*Radius;
	totalVolume /= 0.68;
	float totalRadius = pow(3.0*totalVolume/(4.0*PI), 1.0/3.0);
	GlobeRadius = 2.0*totalRadius;

	// Randomly setting these bodies in the glaobal sphere and setting the initial velosity, inotial force, and mass.
    Damp = 0.5;
	for(int i = 0; i < N; i++)
	{
		test = 0;
		while(test == 0)
		{
			// Get random position.
			randomAngle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			randomAngle2 = ((float)rand()/(float)RAND_MAX)*PI;
			randomRadius = ((float)rand()/(float)RAND_MAX)*GlobeRadius;
			P[i].x = randomRadius*cos(randomAngle1)*sin(randomAngle2);
			P[i].y = randomRadius*sin(randomAngle1)*sin(randomAngle2);
			P[i].z = randomRadius*cos(randomAngle2);
			
			// Making sure the bodies' centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				dx = P[i].x-P[j].x;
				dy = P[i].y-P[j].y;
				dz = P[i].z-P[j].z;
				d = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < Diameter)
				{
					test = 0;
					break;
				}
			}
		}
		V[i].x = 0.0;
		V[i].y = 0.0;
		V[i].z = 0.0;
		
		F[i].x = 0.0;
		F[i].y = 0.0;
		F[i].z = 0.0;
		
		M[i] = 1.0;
	}


	// Because this is a double pointer, or an array of pointers, I think I need the address of each pointer in order
	// to make this work... Might as well try!
	for(int i = 0; i < NumberOfGpus; i++) 
	{
		hipSetDevice(i);
		hipMemcpyAsync(&PGPU[i], P, N*sizeof(float3), hipMemcpyHostToDevice);
		cudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(&VGPU[i], V, N*sizeof(float3), hipMemcpyHostToDevice);
		cudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(&FGPU[i], F, N*sizeof(float3), hipMemcpyHostToDevice);
		cudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(&MGPU[i], M, N*sizeof(float), hipMemcpyHostToDevice);
		cudaErrorCheck(__FILE__, __LINE__);
	}
	
	printf("\n Setup finished.\n");
}

// Position, Velocity, Force, Mass, Gravity, Time Step (sigma thing), Bodies On Gpu, Total Number of Bodies, Shift Over
__global__ void getForces(float3 *p, float3 *v, float3 *f, float *m, float g, float h, int gpuB, int N, int shift)
{
    float dx, dy, dz, d, d2;
    float force_mag;
    
    // Global Index
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    if(i < gpuB)
	{
		i += shift; // Make sure I shift over

		// Zero out forces
		f[i].x = 0.0f;
		f[i].y = 0.0f;
		f[i].z = 0.0f;
		
		for(int j = 0; j < N; j++)
		{
			if(i != j)
			{
				dx = p[j].x-p[i].x; // Find the change in the X position
				dy = p[j].y-p[i].y; // Find the change in the Y position
				dz = p[j].z-p[i].z; // Find the change in the Z position
				d2 = dx*dx + dy*dy + dz*dz; // Find distance squared
				d  = sqrt(d2); // Find distance
				
				// Calculate force 
				force_mag  = (g*m[i]*m[j])/(d2) - (h*m[i]*m[j])/(d2*d2); 
				f[i].x += force_mag*dx/d;
				f[i].y += force_mag*dy/d;
				f[i].z += force_mag*dz/d;
			}
		}
	}
}

// Position, Velocity, Force, Mass, dampening, change in time, time, Bodies On Gpu, Total Number of Bodies, Shift Over
__global__ void moveBodies(float3 *p, float3 *v, float3 *f, float *m, float damp, float dt, float t, int gpuB, int N, int shift)
{
    // Global Index 
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    if(i < gpuB)
	{
		i += shift;
		
		// I thought about optimizing this, but I don't want to accidentally confuse myself...
		if(t == 0.0f) // Initial Time Step
		{
			v[i].x += ((f[i].x-damp*v[i].x)/m[i])*dt/2.0f;
			v[i].y += ((f[i].y-damp*v[i].y)/m[i])*dt/2.0f;
			v[i].z += ((f[i].z-damp*v[i].z)/m[i])*dt/2.0f;
		}
		else // Everything else
		{
			v[i].x += ((f[i].x-damp*v[i].x)/m[i])*dt;
			v[i].y += ((f[i].y-damp*v[i].y)/m[i])*dt;
			v[i].z += ((f[i].z-damp*v[i].z)/m[i])*dt;
		}

		// Update position
		p[i].x += v[i].x*dt;
		p[i].y += v[i].y*dt;
		p[i].z += v[i].z*dt;
	}
}


void nBody()
{
	int    drawCount = 0; 
	float  t = 0.0;
	float dt = 0.0001;
	int shift = BGPU[0];

	while(t < RUN_TIME)
	{
		// Run through all gpus, calculate forces and move bodies!
		for(int device = 0; device < NumberOfGpus; device++)
		{
			hipSetDevice(device);
			getForces<<<GridSizes[device],BlockSize>>>(PGPU[device], VGPU[device], FGPU[device], MGPU[device], G, H, BGPU[device], N, i*shift);
			cudaErrorCheck(__FILE__, __LINE__);
			moveBodies<<<GridSizes[device],BlockSize>>>(PGPU[device], VGPU[device], FGPU[device], MGPU[device], Damp, dt, t, BGPU[device], N, i*shift);
			cudaErrorCheck(__FILE__, __LINE__);
		}
		
		// Sync all devices!
		for(int device = 0; device < NumberOfGpus; device++)
		{
			hipSetDevice(device);
			hipDeviceSynchronize();
			cudaErrorCheck(__FILE__, __LINE__);
		}
		
		// Copying memory between GPUs. You do not need to copy velocities because velocities are only updated with 
		// information from it's own thread.
		for(int device = 0; device < NumberOfGpus; device++)
		{
			hipSetDevice(device);
			for(int otherDevice = 0; otherDevice < NumberOfGpus; otherDevice++)
			{
				if(i != j)
				{
					hipMemcpyAsync(PGPU[otherDevice] + shift*device, PGPU[device] + shift*device, BGPU[device]*sizeof(float3), hipMemcpyDeviceToDevice);
					cudaErrorCheck(__FILE__, __LINE__);
				}
			}	
		}
		
		// Sync all devices!
		for(int device = 0; device < NumberOfGpus; device++)
		{
			hipSetDevice(device);
			hipDeviceSynchronize();
			cudaErrorCheck(__FILE__, __LINE__);
		}

		if(drawCount == DRAW_RATE) 
		{	
			drawPicture();
			drawCount = 0;
		}
		
		t += dt;
		drawCount++;
	}
}


// Free memory
void cleanUpRoom()
{
	// Free memory on CPU
	free(P);
	free(V);
	free(F);
	free(M);

	// Free the bodies
	free(BGPU);
	
	// free GPU pointers first!
	for (int i = 0; i < NumberOfGpus; i++) {
		hipSetDevice(i);
		if(PGPU[i]) hipFree(PGPU[i]);
		if(VGPU[i]) hipFree(VGPU[i]);
		if(FGPU[i]) hipFree(FGPU[i]);
		if(MGPU[i]) hipFree(MGPU[i]);
	}
	
	// Free the host side
	free(PGPU);
	free(VGPU);
	free(FGPU);
	free(MGPU);
}



int main(int argc, char** argv)
{
	setup();
	
	int XWindowSize = 1000;
	int YWindowSize = 1000;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Nbody Two GPUs");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(drawPicture);
	glutIdleFunc(nBody);
	
	float3 eye = {0.0f, 0.0f, 2.0f*GlobeRadius};
	float near = 0.2;
	float far = 5.0*GlobeRadius;
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, near, far);
	glMatrixMode(GL_MODELVIEW);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	gluLookAt(eye.x, eye.y, eye.z, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	
	glutMainLoop();

	// Clean up the room
	cleanUpRoom();
	return 0;
}

#include "hip/hip_runtime.h"
// Name: Conner Homrighaus
// nBody code on multiple GPUs. 
// nvcc HW24.cu -o temp -lglut -lm -lGLU -lGL

/*
 What to do:
 This is some robust N-body code with all the bells and whistles removed. 
 Modify it so it runs on two GPUs.
*/

// Include files
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Defines
#define BLOCK_SIZE 128
#define PI 3.14159265359
#define DRAW_RATE 10

// This is to create a Lennard-Jones type function G/(r^p) - H(r^q). (p < q) p has to be less than q.
// In this code we will keep it a p = 2 and q = 4 problem. The diameter of a body is found using the general
// case so it will be more robust but in the code leaving it as a set 2, 4 problem make the coding much easier.
#define G 10.0f
#define H 10.0f
#define LJP  2.0
#define LJQ  4.0

#define DT 0.0001
#define RUN_TIME 10.0

// Globals
int N;
float3 *P, *V, *F;
float *M; 
float3 *PGPU, *VGPU, *FGPU;
float *MGPU;
float3 *PGPU1, *VGPU1, *FGPU1;
float *MGPU1;
float GlobeRadius, Diameter, Radius;
float Damp;
dim3 BlockSize;
dim3 GridSize;

// Function prototypes
void cudaErrorCheck(const char *, int);
void drawPicture();
void setup();
__global__ void getForces(float3 *, float3 *, float3 *, float *, float, float, int);
__global__ void moveBodies(float3 *, float3 *, float3 *, float *, float, float, float, int);
void nBody();
void cleanUpRoom();
int main(int, char**);

// Error check
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// Visualization
void drawPicture()
{
	int i;
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	hipMemcpyAsync(P, PGPU, N*sizeof(float3), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	glColor3d(1.0,1.0,0.5);
	for(i=0; i<N; i++)
	{
		glPushMatrix();
		glTranslatef(P[i].x, P[i].y, P[i].z);
		glutSolidSphere(Radius,20,20);
		glPopMatrix();
	}
	glutSwapBuffers();
}

// Setup scene
void setup()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	cudaErrorCheck(__FILE__, __LINE__);
	if (deviceCount < 2) {
		fprintf(stderr, "How Dare You, You Scrub! How Dare You Think You Could Run Me Without Two GPUs!\n");
		fprintf(stderr, "Send me ur paypal and I'll CONSIDER buying you a second GPU you scrub. __(ツ)_/¯\n");
		exit(EXIT_FAILURE);
	}

	float randomAngle1, randomAngle2, randomRadius;
	float d, dx, dy, dz;
	int test;
	
	N = 1000;
	BlockSize.x = BLOCK_SIZE; BlockSize.y = 1; BlockSize.z = 1;
	GridSize.x = (N - 1)/BlockSize.x + 1; GridSize.y = 1; GridSize.z = 1;
	Damp = 0.5;

	M = (float*)malloc(N*sizeof(float));
	P = (float3*)malloc(N*sizeof(float3));
	V = (float3*)malloc(N*sizeof(float3));
	F = (float3*)malloc(N*sizeof(float3));

	hipSetDevice(0);
	hipMalloc(&MGPU, N*sizeof(float));
	hipMalloc(&PGPU, N*sizeof(float3));
	hipMalloc(&VGPU, N*sizeof(float3));
	hipMalloc(&FGPU, N*sizeof(float3));

	hipSetDevice(1);
	hipMalloc(&MGPU1, N*sizeof(float));
	hipMalloc(&PGPU1, N*sizeof(float3));
	hipMalloc(&VGPU1, N*sizeof(float3));
	hipMalloc(&FGPU1, N*sizeof(float3));

	Diameter = pow(H/G, 1.0/(LJQ - LJP));
	Radius = Diameter/2.0;
	float totalVolume = float(N)*(4.0/3.0)*PI*Radius*Radius*Radius;
	totalVolume /= 0.68;
	float totalRadius = pow(3.0*totalVolume/(4.0*PI), 1.0/3.0);
	GlobeRadius = 2.0*totalRadius;

	for(int i = 0; i < N; i++)
	{
		test = 0;
		while(test == 0)
		{
			randomAngle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			randomAngle2 = ((float)rand()/(float)RAND_MAX)*PI;
			randomRadius = ((float)rand()/(float)RAND_MAX)*GlobeRadius;
			P[i].x = randomRadius*cos(randomAngle1)*sin(randomAngle2);
			P[i].y = randomRadius*sin(randomAngle1)*sin(randomAngle2);
			P[i].z = randomRadius*cos(randomAngle2);
			test = 1;
			for(int j = 0; j < i; j++)
			{
				dx = P[i].x-P[j].x;
				dy = P[i].y-P[j].y;
				dz = P[i].z-P[j].z;
				d = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < Diameter)
				{
					test = 0; break;
				}
			}
		}
		V[i] = F[i] = make_float3(0.0, 0.0, 0.0);
		M[i] = 1.0;
	}

	hipSetDevice(0);
	hipMemcpyAsync(PGPU, P, N*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpyAsync(VGPU, V, N*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpyAsync(FGPU, F, N*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpyAsync(MGPU, M, N*sizeof(float), hipMemcpyHostToDevice);

	hipSetDevice(1);
	hipMemcpyAsync(PGPU1, P, N*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpyAsync(VGPU1, V, N*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpyAsync(FGPU1, F, N*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpyAsync(MGPU1, M, N*sizeof(float), hipMemcpyHostToDevice);
}

// Compute forces
__global__ void getForces(float3 *p, float3 *v, float3 *f, float *m, float g, float h, int n)
{
	float dx, dy, dz,d,d2;
	float force_mag;
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i < n)
	{
		f[i] = make_float3(0.0f, 0.0f, 0.0f);
		for(int j = 0; j < n; j++)
		{
			if(i != j)
			{
				dx = p[j].x-p[i].x;
				dy = p[j].y-p[i].y;
				dz = p[j].z-p[i].z;
				d2 = dx*dx + dy*dy + dz*dz;
				d  = sqrt(d2);
				force_mag = (g*m[i]*m[j])/(d2) - (h*m[i]*m[j])/(d2*d2);
				f[i].x += force_mag*dx/d;
				f[i].y += force_mag*dy/d;
				f[i].z += force_mag*dz/d;
			}
		}
	}
}

// Move bodies
__global__ void moveBodies(float3 *p, float3 *v, float3 *f, float *m, float damp, float dt, float t, int n)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i < n)
	{
		if(t == 0.0f)
		{
			v[i].x += ((f[i].x-damp*v[i].x)/m[i])*dt/2.0f;
			v[i].y += ((f[i].y-damp*v[i].y)/m[i])*dt/2.0f;
			v[i].z += ((f[i].z-damp*v[i].z)/m[i])*dt/2.0f;
		}
		else
		{
			v[i].x += ((f[i].x-damp*v[i].x)/m[i])*dt;
			v[i].y += ((f[i].y-damp*v[i].y)/m[i])*dt;
			v[i].z += ((f[i].z-damp*v[i].z)/m[i])*dt;
		}
		p[i].x += v[i].x*dt;
		p[i].y += v[i].y*dt;
		p[i].z += v[i].z*dt;
	}
}

// Simulation loop
void nBody()
{
	int drawCount = 0; 
	float t = 0.0;
	float dt = 0.0001;
	int halfN = N/2;

	while(t < RUN_TIME)
	{
		hipSetDevice(0);
		getForces<<<GridSize,BlockSize>>>(PGPU, VGPU, FGPU, MGPU, G, H, N);
		moveBodies<<<GridSize,BlockSize>>>(PGPU, VGPU, FGPU, MGPU, Damp, dt, t, halfN);

		hipSetDevice(1);
		getForces<<<GridSize,BlockSize>>>(PGPU1, VGPU1, FGPU1, MGPU1, G, H, N);
		moveBodies<<<GridSize,BlockSize>>>(PGPU1, VGPU1, FGPU1, MGPU1, Damp, dt, t, N - halfN);

		hipSetDevice(0);
		hipMemcpy(PGPU + halfN, PGPU1 + halfN, (N - halfN)*sizeof(float3), hipMemcpyDeviceToDevice);
		hipMemcpy(VGPU + halfN, VGPU1 + halfN, (N - halfN)*sizeof(float3), hipMemcpyDeviceToDevice);

		if(drawCount == DRAW_RATE) 
		{
			drawPicture();
			drawCount = 0;
		}
		t += dt;
		drawCount++;

		hipSetDevice(0);
		hipDeviceSynchronize();
		hipSetDevice(1);
		hipDeviceSynchronize();
	}
}

// Free memory
void cleanUpRoom()
{
	free(P);
	free(V);
	free(F);
	free(M);
	
	hipSetDevice(0);
	hipFree(PGPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(VGPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(FGPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(MGPU);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipSetDevice(1);
	hipFree(PGPU1);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(VGPU1);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(FGPU1);
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(MGPU1);
	cudaErrorCheck(__FILE__, __LINE__);
}

int main(int argc, char** argv)
{
	setup();
	
	int XWindowSize = 1000;
	int YWindowSize = 1000;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Nbody Two GPUs");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(drawPicture);
	glutIdleFunc(nBody);
	
	float3 eye = {0.0f, 0.0f, 2.0f*GlobeRadius};
	float near = 0.2;
	float far = 5.0*GlobeRadius;
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, near, far);
	glMatrixMode(GL_MODELVIEW);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	gluLookAt(eye.x, eye.y, eye.z, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	
	glutMainLoop();
	cleanUpRoom();
	return 0;
}

